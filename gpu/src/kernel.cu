#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernel.h"
const int BLOCK_SIZE=8;
typedef float (*f_func_ptr)(const float ); 
typedef float (*b_func_ptr)(const float , const float ); 

__global__ void initialize_gradient_(float *grad_bias, float *grad_weights, int M){
	//grad weights has M columns
	int row = blockIdx.x;
	int col = threadIdx.x;	

	if(col == 0){
		grad_bias[row] = 0.0f;
	}
	grad_weights[row*M + col] = 0.0f;
}

__global__ void find_delta_and_transpose(float *a, float *b, float *output, int N, int M){
	int bx = blockIdx.x, by = blockIdx.y;	
	int row = threadIdx.y, col = threadIdx.x;
	int row_in_matrix = by*BLOCK_SIZE+row;
	int col_in_matrix = bx*BLOCK_SIZE+col;
	int index_in_vector = row_in_matrix*M + col_in_matrix;
	int transpose_index_in_vector = col_in_matrix*N + row_in_matrix;
	if(row_in_matrix < N && col_in_matrix< M){
		output[transpose_index_in_vector] = a[index_in_vector] - b[index_in_vector];

		// //debug
		// if(index_in_vector == 0){
		// 	for(int i=0;i < N; ++i)printf("find_delta: %f ", b[i]); 
		// }

	}

}

__global__ void dev_vec_matmul(const float *dev_a, const float *dev_b, float *dev_output, int N, int M){
	//a is NxM
	int ROW = threadIdx.x;
	float temp_value=0;

	//// debug
	//if(ROW == 0){
		//printf("%f\n", activation_func_map[activ_func_ind](-1.0f));
	//}

	for(size_t i=0; i < M; ++i){
		temp_value += dev_a[ROW*M + i]*dev_b[i];
	}
	dev_output[ROW] = temp_value;
}

__global__ void gemm(const float *a, const float *b, const float *c, float *output, int N, int M, int K){
	//a is NxK
	//b is KxM
	//c is Nx1
	int bx = blockIdx.x, by = blockIdx.y;	
	int row = threadIdx.y, col = threadIdx.x;
	//loop to find the sub-matrix of output
	//iterates through sub-matrices of A and B to copy to shared memory
	int row_in_matrix = by*BLOCK_SIZE+row;
	int col_in_matrix = bx*BLOCK_SIZE+col;
	if(row_in_matrix < N && col_in_matrix< M){
		float temp_value=0;
		for(int i=0; i < K; i+=BLOCK_SIZE){
			__shared__ float A[BLOCK_SIZE][BLOCK_SIZE], B[BLOCK_SIZE][BLOCK_SIZE]; 

			//copying to shared memory
			for(int j=0; j < BLOCK_SIZE; ++j){
				for(int k=0; k < BLOCK_SIZE; ++k){
					// this is not good, maybe BLOCK_SIZE can change depending upon the network arch
					// should i move this if outside, like (by+1)*BLOCK_SIZE < N? maybe wrap divergence may happen then
					if(by*BLOCK_SIZE+j < N && i+k < K){
						A[j][k] = a[(by*BLOCK_SIZE+j)*K + i+k]; //i+k -> column, by*BLOCK_SIZE+j -> row 
					}else{
						A[j][k] = 0;
					}
					if(i+j < K && bx*BLOCK_SIZE+k < M){
						B[j][k] = b[(i+j)*M + bx*BLOCK_SIZE+k]; //bx*BLOCK_SIZE+k -> column, i+j -> row 
					}else{
						B[j][k] = 0;
					}
				}
			}


			//wait for completion
			__syncthreads();

			//multiply and add
			for(int j=0; j < BLOCK_SIZE; ++j){
				temp_value += A[row][j]*B[j][col];
			}

			__syncthreads();
			

		}

		// //debug
		// int index = row_in_matrix*M + col_in_matrix;
		// if(index == 0){
		// 	printf("gemm : %f\n", temp_value+c[0]);
		// }

		output[(row_in_matrix)*M + col_in_matrix] = temp_value+c[row_in_matrix];
	}
}

__global__ void matmul(const float *a, const float *b, float *output, int N, int M, int K){
	//a is NxK
	//b is KxM
	//c is NxM
	int bx = blockIdx.x, by = blockIdx.y;	
	int row = threadIdx.y, col = threadIdx.x;
	//loop to find the sub-matrix of output
	//iterates through sub-matrices of A and B to copy to shared memory
	int row_in_matrix = by*BLOCK_SIZE+row;
	int col_in_matrix = bx*BLOCK_SIZE+col;
	if(row_in_matrix < N && col_in_matrix< M){
		float temp_value=0;
		for(int i=0; i < K; i+=BLOCK_SIZE){
			__shared__ float A[BLOCK_SIZE][BLOCK_SIZE], B[BLOCK_SIZE][BLOCK_SIZE]; 

			//copying to shared memory
			for(int j=0; j < BLOCK_SIZE; ++j){
				for(int k=0; k < BLOCK_SIZE; ++k){
					if(by*BLOCK_SIZE+j < N && i+k < K){
						A[j][k] = a[(by*BLOCK_SIZE+j)*K + i+k]; //i+k -> column, by*BLOCK_SIZE+j -> row 
					}else{
						A[j][k] = 0;
					}
					if(i+j < K && bx*BLOCK_SIZE+k < M){
						B[j][k] = b[(i+j)*M + bx*BLOCK_SIZE+k]; //bx*BLOCK_SIZE+k -> column, i+j -> row 
					}else{
						B[j][k] = 0;
					}
				}
			}

			////debug
			//if(row == 0 && col == 0 && bx == 0 && by == 0){
				//printf("kernel A : %f\n", A[0][0]);
				//printf("kernel B : %f\n", B[0][0]);
			//}


			//wait for completion
			__syncthreads();

			//multiply and add
			for(int j=0; j < BLOCK_SIZE; ++j){
				temp_value += A[row][j]*B[j][col];
			}

			__syncthreads();


		}

		////debug
		//if(row == 0 && col == 0 && bx == 0 && by == 0){
			//printf("kernel : %f\n", (*function)(-0.5));
		//}

		//int transposed_index =(col_in_matrix)*N + row_in_matrix;
		output[(row_in_matrix)*M + col_in_matrix] = temp_value;
	}
}

__global__ void update_bias(float *dev_bias, float *layer_delta, int N, int M, float learning_rate){ 
	/*
	* bias will be (M, 1) dim
	* N is no of samples
	* M is rows of the layer 
	*/
	int COL = threadIdx.x;
	float temp_value=0;

	//// debug
	//if(ROW == 0){
		//printf("%f\n", activation_func_map[activ_func_ind](-1.0f));
	//}
	for(size_t i=0; i < N; ++i){
		temp_value += layer_delta[i*M + COL];
	}
	dev_bias[COL] += learning_rate*temp_value/N;
}
__global__ void update_weights(float *dev_weights, float *layer_output, float *layer_delta, int N, int M, int K, float learning_rate){
/*
 * will multiply layer_output and layer_delta together
 * transpose(weights) += layer_output*layer_delta*learning_rate
 *
 * layer_outputs is NxK
 * layer_deltas is KxM
 * K is no of samples
 * 
 * assume a layer is (4,2). i.e. 4 rows and 2 cols
 * updating the weights of this layer requires the following matrices
 * layer_output ---> (2,no_of_samples)
 * layer_deltas ---> (no_of_samples, 4)
 * 
 * doing layer_output*layer_deltas gives a (2,4) matrix. 
 * thus we need to transpose it.
 */
	int bx = blockIdx.x, by = blockIdx.y;	
	int row = threadIdx.y, col = threadIdx.x;
	int row_in_matrix = by*BLOCK_SIZE+row;
	int col_in_matrix = bx*BLOCK_SIZE+col;
	if(row_in_matrix < N && col_in_matrix< M){
		float temp_value = 0;
		for(int i=0; i < K; i+=BLOCK_SIZE){
			__shared__ float A[BLOCK_SIZE][BLOCK_SIZE], B[BLOCK_SIZE][BLOCK_SIZE]; 

			//copying to shared memory
			for(int j=0; j < BLOCK_SIZE; ++j){
				for(int k=0; k < BLOCK_SIZE; ++k){
					if(by*BLOCK_SIZE+j < N && i+k < K){
						A[j][k] = layer_output[(by*BLOCK_SIZE+j)*K + i+k]; //i+k -> column, by*BLOCK_SIZE+j -> row 
					}else{
						A[j][k] = 0;
					}
					if(i+j < K && bx*BLOCK_SIZE+k < M){
						B[j][k] = layer_delta[(i+j)*M + bx*BLOCK_SIZE+k]; //bx*BLOCK_SIZE+k -> column, i+j -> row 
					}else{
						B[j][k] = 0;
					}
				}
			}

			//wait for completion
			__syncthreads();

			//multiply and add
			for(int j=0; j < BLOCK_SIZE; ++j){
				temp_value += A[row][j]*B[j][col];
			}

			__syncthreads();
		}

		//transpose added

		////debug
		//if(row == 0 && col == 0 && bx == 0 && by == 0){
			//printf("%f\n", A[0][0]);
		//}
		//printf("%f\n\n", learning_rate*temp_value);

		//NOTE : both these statements work for xor at least, wtf
		// dev_weights[row_in_matrix*M + col_in_matrix] += learning_rate*temp_value/K;
		dev_weights[col_in_matrix*N + row_in_matrix] += learning_rate*temp_value/K;
	}
}

__global__ void rmsprop_update_bias_(float *dev_bias, float *layer_delta, float *gradient_sum, int N, int M, float learning_rate, float beta){ 
	// layer_delta is NxM
	int COL = threadIdx.x;
	float temp_value=0;

	for(size_t i=0; i < N; ++i){
		temp_value += layer_delta[i*M + COL];
	}

	// //debug
	// if(COL == 0){
	// 	printf("rmsprop_bias_kernel : %f\n", layer_delta[0]);
	// }

	gradient_sum[COL] = gradient_sum[COL]*beta + (1-beta)*temp_value*temp_value;
	dev_bias[COL] += learning_rate*temp_value/(1e-5 + sqrt(gradient_sum[COL]));
	// NOTE : MAX SCAMM
	// dev_bias[COL] += learning_rate*temp_value/(1 + sqrt(gradient_sum[COL]));
}

__global__ void rmsprop_update_weights_(float *dev_weights, float *layer_output, float *layer_delta, float *gradient_sum, int N, int M, int K, float learning_rate, float beta){
/*
 * will multiply layer_output and layer_delta together
 * transpose(weights) += layer_output*layer_delta*learning_rate/root(gradient_sum)
 *
 * layer_outputs is NxK
 * layer_deltas is KxM
 * K is no of samples
 * 
 * assume a layer is (4,2). i.e. 4 rows and 2 cols
 * updating the weights of this layer requires the following matrices
 * layer_output ---> (2,no_of_samples)
 * layer_deltas ---> (no_of_samples, 4)
 * 
 * doing layer_output*layer_deltas gives a (2,4) matrix. 
 * thus we need to transpose it.
 */
	int bx = blockIdx.x, by = blockIdx.y;	
	int row = threadIdx.y, col = threadIdx.x;
	int row_in_matrix = by*BLOCK_SIZE+row;
	int col_in_matrix = bx*BLOCK_SIZE+col;
	// int index_in_matrix = row_in_matrix*M + col_in_matrix;
	if(row_in_matrix < N && col_in_matrix< M){
		float temp_value = 0;
		for(int i=0; i < K; i+=BLOCK_SIZE){
			__shared__ float A[BLOCK_SIZE][BLOCK_SIZE], B[BLOCK_SIZE][BLOCK_SIZE]; 

			//copying to shared memory
			for(int j=0; j < BLOCK_SIZE; ++j){
				for(int k=0; k < BLOCK_SIZE; ++k){
					if(by*BLOCK_SIZE+j < N && i+k < K){
						A[j][k] = layer_output[(by*BLOCK_SIZE+j)*K + i+k]; //i+k -> column, by*BLOCK_SIZE+j -> row 
					}else{
						A[j][k] = 0;
					}
					if(i+j < K && bx*BLOCK_SIZE+k < M){
						B[j][k] = layer_delta[(i+j)*M + bx*BLOCK_SIZE+k]; //bx*BLOCK_SIZE+k -> column, i+j -> row 
					}else{
						B[j][k] = 0;
					}

				}
			}

			//wait for completion
			__syncthreads();

			//multiply and add
			for(int j=0; j < BLOCK_SIZE; ++j){
				temp_value += A[row][j]*B[j][col];
			}

			__syncthreads();
		}

		////debug
		//if(row == 0 && col == 0 && bx == 0 && by == 0){
			//printf("%f\n", A[0][0]);
		//}
		//printf("%f\n\n", learning_rate*temp_value);


		//should this be transposed???
		int transposed_ind = col_in_matrix*N + row_in_matrix;
		gradient_sum[transposed_ind] = beta*gradient_sum[transposed_ind] + (1-beta)*temp_value*temp_value;
		dev_weights[transposed_ind] += learning_rate*temp_value/(1e-5 + sqrt(gradient_sum[transposed_ind]));
		//NOTE : MAX SCAMM
		// dev_weights[transposed_ind] += learning_rate*temp_value/(1 + sqrt(gradient_sum[transposed_ind]));
	}
}
