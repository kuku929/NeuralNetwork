#include "hip/hip_runtime.h"
#include "basic_matrix.h"
#include "dev_vector.h"
#include "loss.h"
using namespace nnet;

#if defined(DEBUG)
#define shared_dev_vector(a, b) shared_dev_vector(a, b, __LINE__, __FILE_NAME__)
#define set(a, b) set(a, b, __LINE__, __FILE_NAME__)
#endif

template <typename loss_func>
__global__ void find_loss_T(float *prediction, float *actual, float *output, size_t no_of_samples,
                            size_t size, loss_func loss)
{
    int col = threadIdx.x;
    float temp;
    for (int i = col; i <= (size - 1) * no_of_samples + col; i += no_of_samples)
    {
        temp += loss(prediction[i], actual[i]);
    }
    output[col] = temp;
}

template <typename loss_func>
__global__ void find_loss_(float *prediction, float *actual, float *output, size_t no_of_samples,
                           size_t size, loss_func loss)
{
    /*
     * am i tripping or this function is wrong??
     * true output has (size, no_of_samples) dim
     * right??
     */
    int row = threadIdx.x;
    float temp;
    for (int i = row * size; i < (row + 1) * size; ++i)
    {
        temp += loss(prediction[i], actual[i]);
    }
    output[row] = temp;
}

template <typename loss_func>
__global__ void find_loss_derivative_(float *prediction, float *actual, float *output,
                                      size_t no_of_samples, size_t size, loss_func loss_deriv)
{
    // NOTE : i am transposing right now, but need to think of a better way. row
    // is the row in input BROOOOOOOOOOOOOO THIS IS SHIT I NEED TO DO SOMETHING!
    // figuring out the correct dimensions is a PAIN IN THE ASS
    // think of a way so that my code is dimension independant or they are
    // implicitly handled OVER AND OUT -- graph theory tmrw ;<
    // Damn, I had a panic attack right here,. futures bad-08/09/24

    int row = threadIdx.x;
    int col = blockIdx.x;
    int index_in_matrix = row * no_of_samples + col;
    // transposing
    output[col * size + row] = loss_deriv(prediction[index_in_matrix], actual[index_in_matrix]);
}

dev_ptr MSELoss::find_loss(dev_vector<float> &prediction, dev_vector<float> &actual,
                           size_t no_of_samples)
{
    // output should be of size no_of_samples,1
    dev_ptr output = shared_dev_vector(no_of_samples, 1);
    dim3 dim_block(no_of_samples, 1);
    find_loss_T<<<1, dim_block>>>(prediction.data(), actual.data(), output->data(), no_of_samples,
                                  size, mse_loss_func_);
    hipDeviceSynchronize();
    return output;
}

dev_ptr MSELoss::loss_derivative(dev_vector<float> &prediction, dev_vector<float> &actual,
                                 size_t no_of_samples)
{
    auto output = shared_dev_vector(no_of_samples, size);
    dim3 dim_block(size, 1);
    dim3 dim_grid(no_of_samples, 1);
    find_loss_derivative_<<<dim_grid, dim_block>>>(prediction.data(), actual.data(), output->data(),
                                                   no_of_samples, size, mse_loss_der_);
    hipDeviceSynchronize();
    return output;
}

dev_ptr CrossEntropyLoss::find_loss(dev_vector<float> &prediction, dev_vector<float> &actual,
                                    size_t no_of_samples)
{
    dev_ptr output = shared_dev_vector(no_of_samples, 1);
    dim3 dim_block(size, 1);
    dim3 dim_grid(no_of_samples, 1);
    find_loss_T<<<1, dim_block>>>(prediction.data(), actual.data(), output->data(), no_of_samples,
                                  size, cross_entropy_loss_func_);
    hipDeviceSynchronize();
    return output;
}

dev_ptr CrossEntropyLoss::loss_derivative(dev_vector<float> &prediction, dev_vector<float> &actual,
                                          size_t no_of_samples)
{
    auto output = shared_dev_vector(no_of_samples, size);
    dim3 dim_block(size, 1);
    dim3 dim_grid(no_of_samples, 1);
    find_loss_derivative_<<<dim_grid, dim_block>>>(prediction.data(), actual.data(), output->data(),
                                                   no_of_samples, size, cross_entropy_loss_der_);
    hipDeviceSynchronize();
    return output;

    ////debug
    // basic_matrix<float> o(size, no_of_samples);
    // hipMemcpy(o.data(), prediction_softmax.begin(), sizeof(float) * o.size,
    //            hipMemcpyDeviceToHost);
    // o.show();
}
